/*
  We need 3 steps to make CPU to GPU:
  1) Parallelized the vector function
  2) All CPU move to GPU and execute and move back to CPU
  3) Finally modify the vectorAdd to modify the function in GPU
*/
#include<studio.h>
#define SIZE 1024
/*
  parallelized the vector
  it tells the compiler that it will execute in GPU
*/
_global_void vectorAdd(int *a, int *b, int *c, int n){
  int i = threadId.x; // needed for thread
  if(i < n)
    c[i] = a[i] + b[i];
} // _global_void vector
int main(){
  int *a, *b, *c;
  int *d_a, *d_b, *d_c; // variables for GPU
  // for CPU Memory Allocation
  a = (int*)malloc(SIZE*sizeof(int));
  b = (int*)malloc(SIZE*sizeof(int));
  c = (int*)malloc(SIZE*sizeof(int));
  // for GPU Memory Allocation, not a back up pointer, use address &
  hipMalloc( &d_a, SIZE*sizeof(int));
  hipMalloc( &d_b, SIZE*sizeof(int));
  hipMalloc( &d_c, SIZE*sizeof(int));
  // Copy the data CPU to the GPU
  //  cudaMemCpy( destination, source, SIZE, direction of the data);
  cudaMemCpy( d_a, a, SIZE*sizeof(int), cudaMemCpyHostToDevice);
  cudaMemCpy( d_b, b, SIZE*sizeof(int), cudaMemCpyHostToDevice);
  cudaMemCpy( d_c, c, SIZE*sizeof(int), cudaMemCpyHostToDevice);
  for(int i = 0; i < SIZE; ++i){
    a[i] = i;
    b[i] = i;
    c[i] = 0;
  }
  // Syntax change for CUDA
  // vectorAdd <<< (no of block, Size of block) >>> (d_a, d_b, d_c, SIZE);
  vectorAdd <<< (1, SIZE) >>> (d_a, d_b, d_c, SIZE);
  // Copy the data GPU to the CPU
  cudaMemCpy( destination, source, SIZE, direction of the data);
  cudaMemCpy( c, d_c, SIZE*sizeof(int), cudaMemCpyDeviceToHost);
  for(int i = 0; i < 10; ++i)
    printf("c[%d] = %d\n", i, c[i]);
  // making free the CPU
  free(a);
  free(b);
  free(c);
  // making free the GPU
  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_c);
  return 0;
}
