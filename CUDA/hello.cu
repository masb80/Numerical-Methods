
#include <hip/hip_runtime.h>
#include <iostream>
using namespace std;

const int N = 16;
const int CORES = 16;

/* this is the GPU kernel function */
__global__ void hello(char* s) {
  /* blockIdx is a struct containing our block id
     if this this is a one-dimensional kernel, then x is the block id
     y and z are also available for 2 or 3 dimensional kernels */

  /* capitalize the string by subtracting 32 from each lowercase letter */
  if ((s[blockIdx.x] >= 'a') && (s[blockIdx.x] <= 'z')) {
    s[blockIdx.x] -= 32;
  }
}

/* the main function begins running on the CPU */
int main( ) {
  /* this is the string data - it is 'hello world', in lower-case */
  char cpu_string[N] = "hello world!";

  /* allocate space on the GPU for the string */
  char* gpu_string;
  hipMalloc((void**) &gpu_string, N * sizeof(char));

  /* send the character array to the GPU */
  hipMemcpy(gpu_string, cpu_string, N * sizeof(char), hipMemcpyHostToDevice);
  
  /* invoke the GPU to run the kernel in parallel
     we specify CORES cores which each run once */
  hello<<<CORES, 1>>>(gpu_string);

  /* copy the string back from the GPU to the CPU */
  hipMemcpy(cpu_string, gpu_string, N * sizeof(char), hipMemcpyDeviceToHost);
  
  /* free the memory we allocated on the GPU */
  hipFree(gpu_string);
  
  /* print the string we got back from the GPU */
  cout << cpu_string << endl;

  return 0;
}